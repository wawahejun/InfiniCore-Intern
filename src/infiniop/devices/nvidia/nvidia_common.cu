#include "nvidia_handle.cuh"

namespace device {

namespace nvidia {

Handle::Handle(infiniDevice_t device, int device_id)
    : InfiniopHandle{device, device_id},
      _internal(std::make_shared<Handle::Internal>(device_id)) {}

auto Handle::internal() const -> const std::shared_ptr<Internal> & {
    return _internal;
}

Handle::Internal::Internal(int device_id) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);
    _warp_size = prop.warpSize;
    _max_threads_per_block = prop.maxThreadsPerBlock;
    _block_size[0] = prop.maxThreadsDim[0];
    _block_size[1] = prop.maxThreadsDim[1];
    _block_size[2] = prop.maxThreadsDim[2];
    _grid_size[0] = prop.maxGridSize[0];
    _grid_size[1] = prop.maxGridSize[1];
    _grid_size[2] = prop.maxGridSize[2];
}

infiniStatus_t Handle::Internal::useCublas(hipStream_t stream, const Fn<hipblasHandle_t> &f) const {
    auto handle = blas_handles.pop();
    if (!handle) {
        CHECK_CUBLAS(hipblasCreate(&(*handle)));
    }
    CHECK_CUBLAS(hipblasSetStream(*handle, stream));
    CHECK_STATUS(f(*handle));
    blas_handles.push(std::move(*handle));
    return INFINI_STATUS_SUCCESS;
}

#ifdef ENABLE_CUDNN_API
infiniStatus_t Handle::Internal::useCudnn(hipStream_t stream, const Fn<hipdnnHandle_t> &f) const {
    auto handle = dnn_handles.pop();
    if (!handle) {
        CHECK_CUDNN(hipdnnCreate(&(*handle)));
    }
    CHECK_CUDNN(hipdnnSetStream(*handle, stream));
    CHECK_STATUS(f(*handle));
    dnn_handles.push(std::move(*handle));
    return INFINI_STATUS_SUCCESS;
}
#endif

int Handle::Internal::warpSize() const { return _warp_size; }
int Handle::Internal::maxThreadsPerBlock() const { return _max_threads_per_block; }
int Handle::Internal::blockSizeX() const { return _block_size[0]; }
int Handle::Internal::blockSizeY() const { return _block_size[1]; }
int Handle::Internal::blockSizeZ() const { return _block_size[2]; }
int Handle::Internal::gridSizeX() const { return _grid_size[0]; }
int Handle::Internal::gridSizeY() const { return _grid_size[1]; }
int Handle::Internal::gridSizeZ() const { return _grid_size[2]; }

#ifdef ENABLE_CUDNN_API
hipdnnDataType_t getCudnnDtype(infiniDtype_t dt) {
    switch (dt) {
    case INFINI_DTYPE_F16:
        return HIPDNN_DATA_HALF;
    case INFINI_DTYPE_F32:
        return HIPDNN_DATA_FLOAT;
    case INFINI_DTYPE_F64:
        return HIPDNN_DATA_DOUBLE;
    case INFINI_DTYPE_BF16:
        return CUDNN_DATA_BFLOAT16;
    case INFINI_DTYPE_I8:
        return HIPDNN_DATA_INT8;
    case INFINI_DTYPE_I32:
        return HIPDNN_DATA_INT32;
#ifndef ENABLE_ILUVATAR_API
    case INFINI_DTYPE_I64:
        return CUDNN_DATA_INT64;
#endif
    case INFINI_DTYPE_U8:
        return CUDNN_DATA_UINT8;
    default:
        return HIPDNN_DATA_FLOAT;
    }
}
#endif

infiniStatus_t Handle::create(InfiniopHandle **handle_ptr, int device_id) {
    *handle_ptr = new Handle(INFINI_DEVICE_NVIDIA, device_id);
    return INFINI_STATUS_SUCCESS;
}

} // namespace nvidia

namespace iluvatar {

Handle::Handle(int device_id)
    : nvidia::Handle(INFINI_DEVICE_ILUVATAR, device_id) {}

infiniStatus_t Handle::create(InfiniopHandle **handle_ptr, int device_id) {
    *handle_ptr = new Handle(device_id);
    return INFINI_STATUS_SUCCESS;
}

} // namespace iluvatar

} // namespace device
