#include "../../../devices/nvidia/nvidia_handle.cuh"
#include "gemm_nvidia.cuh"

namespace op::gemm::nvidia {

struct Descriptor::Opaque {
    std::shared_ptr<device::nvidia::Handle::Internal> internal;
};

Descriptor::~Descriptor() {
    delete _opaque;
}

infiniStatus_t Descriptor::create(
    infiniopHandle_t handle_,
    Descriptor **desc_ptr,
    infiniopTensorDescriptor_t c_desc,
    infiniopTensorDescriptor_t a_desc,
    infiniopTensorDescriptor_t b_desc) {
    auto handle = reinterpret_cast<device::nvidia::Handle *>(handle_);
    auto dtype = c_desc->dtype();

    CHECK_DTYPE(dtype, INFINI_DTYPE_F16, INFINI_DTYPE_F32, INFINI_DTYPE_BF16);

    auto result = MatmulInfo::create(c_desc, a_desc, b_desc, MatrixLayout::COL_MAJOR);
    CHECK_RESULT(result);

    *desc_ptr = new Descriptor(
        dtype, result.take(), 0,
        new Opaque{handle->internal()},
        handle->device, handle->device_id);
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t Descriptor::calculate(
    void *workspace,
    size_t workspace_size,
    void *c,
    float beta,
    const void *a,
    const void *b,
    float alpha,
    void *stream) const {

    hipDataType a_type, b_type, c_type;
#ifdef ENABLE_ILUVATAR_API
    hipDataType compute_type;
#else
    hipblasComputeType_t compute_type;
#endif

    switch (_dtype) {
    case INFINI_DTYPE_F16:
        a_type = b_type = c_type = HIP_R_16F;
#ifdef ENABLE_ILUVATAR_API
        compute_type = HIP_R_32F;
#else
        compute_type = HIPBLAS_COMPUTE_32F;
#endif
        break;
    case INFINI_DTYPE_BF16:
        a_type = b_type = c_type = HIP_R_16BF;
#ifdef ENABLE_ILUVATAR_API
        compute_type = HIP_R_32F;
#else
        compute_type = HIPBLAS_COMPUTE_32F;
#endif
        break;
    case INFINI_DTYPE_F32:
        a_type = b_type = c_type = HIP_R_32F;
#if defined ENABLE_ILUVATAR_API
        compute_type = HIP_R_32F;
#elif defined ENABLE_SUGON_CUDA_API
        compute_type = HIPBLAS_COMPUTE_32F;
#else
        compute_type = HIPBLAS_COMPUTE_32F_FAST_TF32;
#endif
        break;

    default:
        return INFINI_STATUS_BAD_TENSOR_DTYPE;
    }

    if (_info.is_transed) {
        std::swap(a, b);
    }

    auto op_a = _info.a_matrix.row_stride == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    auto op_b = _info.b_matrix.row_stride == 1 ? HIPBLAS_OP_N : HIPBLAS_OP_T;

    CHECK_STATUS(_opaque->internal->useCublas(
        (hipStream_t)stream,
        [&](hipblasHandle_t handle) {
            CHECK_CUBLAS(
                hipblasGemmStridedBatchedEx(
                    handle,
                    op_a,
                    op_b,
                    static_cast<int>(_info.m),
                    static_cast<int>(_info.n),
                    static_cast<int>(_info.k),
                    &alpha,
                    a,
                    a_type,
                    static_cast<int>(_info.a_matrix.ld()),
                    _info.a_matrix.stride,
                    b,
                    b_type,
                    static_cast<int>(_info.b_matrix.ld()),
                    _info.b_matrix.stride,
                    &beta,
                    c,
                    c_type,
                    static_cast<int>(_info.c_matrix.ld()),
                    _info.c_matrix.stride,
                    static_cast<int>(_info.batch),
                    compute_type,
                    CUBLAS_GEMM_DEFAULT_TENSOR_OP));
            return INFINI_STATUS_SUCCESS;
        }));
    return INFINI_STATUS_SUCCESS;
}

} // namespace op::gemm::nvidia
