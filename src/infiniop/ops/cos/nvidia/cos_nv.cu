#include "hip/hip_runtime.h"
#include "../../../elementwise/nvidia/elementwise_nvidia.cuh"

#include "../cuda/kernel.cuh"
#include "cos_nv.cuh"

namespace op::cos::nvidia {

Descriptor::~Descriptor() = default;

infiniStatus_t Descriptor::create(
    infiniopHandle_t handle_,
    Descriptor **desc_ptr,
    infiniopTensorDescriptor_t out_desc,
    std::vector<infiniopTensorDescriptor_t> input_desc_vec) {

    auto handle = reinterpret_cast<device::nvidia::Handle *>(handle_);
    auto dtype = out_desc->dtype();

    const auto &x_desc = input_desc_vec.at(0);
    const auto &y_shape = out_desc->shape();
    const auto &x_shape = x_desc->shape();

    CHECK_DTYPE(dtype, INFINI_DTYPE_F16, INFINI_DTYPE_F32, INFINI_DTYPE_BF16);

    CHECK_SAME_SHAPE(y_shape, x_shape);

    // create CUDA elementwise descriptor
    CREATE_ELEMENTWISE_CUDA_DESCRIPTOR(handle, dtype, out_desc, input_desc_vec)

    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t Descriptor::calculate(
    void *workspace,
    size_t workspace_size,
    void *output,
    std::vector<const void *> inputs,
    void *stream) const {

    if (workspace_size < _workspace_size) {
        return INFINI_STATUS_INSUFFICIENT_WORKSPACE;
    }

    switch (_dtype) {
    case INFINI_DTYPE_F16:
        return _device_info->calculate<256, cuda::CosOp, half>(_info, workspace, output, inputs, stream);
    case INFINI_DTYPE_BF16:
        return _device_info->calculate<256, cuda::CosOp, cuda_bfloat16>(_info, workspace, output, inputs, stream);
    case INFINI_DTYPE_F32:
        return _device_info->calculate<256, cuda::CosOp, float>(_info, workspace, output, inputs, stream);
    default:
        return INFINI_STATUS_BAD_TENSOR_DTYPE;
    }

    return INFINI_STATUS_SUCCESS;
}
} // namespace op::cos::nvidia