#include "hip/hip_runtime.h"
#include "../../../devices/nvidia/nvidia_common.cuh"
#include "reduce_mean_nvidia.cuh"

#include "../../../devices/nvidia/nvidia_kernel_common.cuh"
#include <cub/block/block_reduce.cuh>

#include "../../../reduce/cuda/reduce.cuh"
#include "../../../../utils.h"
#include "infinicore.h"

template <unsigned int BLOCK_SIZE, typename Tdata>
INFINIOP_CUDA_KERNEL reduceMeanKernel(
    Tdata *__restrict__ output,
    const Tdata *__restrict__ input,
    size_t num_reductions,
    size_t reduce_size,
    ptrdiff_t reduce_stride,
    const size_t *input_shape,
    const ptrdiff_t *input_strides,
    const ptrdiff_t *output_strides,
    size_t ndim,
    size_t reduce_dim) {
    
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_reductions) {
        // Calculate the input and output offsets for this reduction
        size_t input_offset = 0;
        size_t output_offset = 0;
        size_t temp_idx = idx;
        
        for (size_t i = 0; i < ndim; i++) {
            if (i != reduce_dim) {
                size_t coord = temp_idx % input_shape[i];
                temp_idx /= input_shape[i];
                input_offset += coord * input_strides[i];
                output_offset += coord * output_strides[i];
            }
        }
        
        // Find the starting position for this reduction
        const Tdata *input_ptr = input + input_offset;
        
        // Calculate the sum
        if constexpr (std::is_same_v<Tdata, fp16_t>) {
            // Use float accumulation for better precision, similar to BF16
            float sum = 0.0f;
            for (size_t i = 0; i < reduce_size; i++) {
                __half val = *reinterpret_cast<const __half*>(&input_ptr[i * reduce_stride]);
                sum += __half2float(val);
            }
            float mean = sum / static_cast<float>(reduce_size);
            __half f16_mean = __float2half(mean);
            output[output_offset] = *reinterpret_cast<const Tdata*>(&f16_mean);
        } else if constexpr (std::is_same_v<Tdata, bf16_t>) {
            float sum = 0.0f;
            for (size_t i = 0; i < reduce_size; i++) {
                __hip_bfloat16 val = *reinterpret_cast<const __hip_bfloat16*>(&input_ptr[i * reduce_stride]);
                sum += __bfloat162float(val);
            }
            float mean = sum / static_cast<float>(reduce_size);
            __hip_bfloat16 bf16_mean = __float2bfloat16(mean);
            output[output_offset] = *reinterpret_cast<const Tdata*>(&bf16_mean);
        } else {
            float sum = 0.0f;
            for (size_t i = 0; i < reduce_size; i++) {
                sum += input_ptr[i * reduce_stride];
            }
            output[output_offset] = sum / static_cast<float>(reduce_size);
        }
    }
}

template <typename T>
infiniStatus_t launchReduceMeanKernel(
    const op::reduce_mean::nvidia::ReduceMeanInfo &info,
    void *output,
    const void *input,
    hipStream_t stream) {
    
    const int BLOCK_SIZE = 256;
    
    // Calculate number of reductions needed
    size_t num_reductions = info.output_size;
    size_t reduce_size = info.input_shape[info.reduce_dim];
    ptrdiff_t reduce_stride = info.input_strides[info.reduce_dim];
    
    // Allocate device memory for shape and strides
    size_t *d_input_shape;
    ptrdiff_t *d_input_strides, *d_output_strides;
    
    hipMalloc(&d_input_shape, info.ndim * sizeof(size_t));
    hipMalloc(&d_input_strides, info.ndim * sizeof(ptrdiff_t));
    hipMalloc(&d_output_strides, info.ndim * sizeof(ptrdiff_t));
    
    hipMemcpy(d_input_shape, info.input_shape.data(), info.ndim * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(d_input_strides, info.input_strides.data(), info.ndim * sizeof(ptrdiff_t), hipMemcpyHostToDevice);
    hipMemcpy(d_output_strides, info.output_strides.data(), info.ndim * sizeof(ptrdiff_t), hipMemcpyHostToDevice);
    
    // Calculate grid size
    size_t grid_size = (num_reductions + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    // Launch kernel
    reduceMeanKernel<BLOCK_SIZE, T><<<grid_size, BLOCK_SIZE, 0, stream>>>(
        static_cast<T*>(output),
        static_cast<const T*>(input),
        num_reductions,
        reduce_size,
        reduce_stride,
        d_input_shape,
        d_input_strides,
        d_output_strides,
        info.ndim,
        info.reduce_dim
    );
    
    // Clean up device memory
    hipFree(d_input_shape);
    hipFree(d_input_strides);
    hipFree(d_output_strides);
    
    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        return INFINI_STATUS_INTERNAL_ERROR;
    }
    
    return INFINI_STATUS_SUCCESS;
}

namespace op::reduce_mean::nvidia {

infiniStatus_t Descriptor::create(
    infiniopHandle_t handle,
    Descriptor **desc_ptr,
    infiniopTensorDescriptor_t output_desc,
    infiniopTensorDescriptor_t input_desc,
    size_t dim) {
    
    // Validate inputs
    if (!input_desc || !output_desc) {
        return INFINI_STATUS_BAD_PARAM;
    }
    
    if (dim >= input_desc->ndim()) {
        return INFINI_STATUS_BAD_PARAM;
    }
    
    // Check dtype support
    if (input_desc->dtype() != INFINI_DTYPE_F32 && 
        input_desc->dtype() != INFINI_DTYPE_F16 && 
        input_desc->dtype() != INFINI_DTYPE_BF16) {
        return INFINI_STATUS_BAD_TENSOR_DTYPE;
    }
    
    // Create info
    ReduceMeanInfo info;
    info.ndim = input_desc->ndim();
    info.reduce_dim = dim;
    info.dtype = input_desc->dtype();
    
    // Copy shapes and strides
    info.input_shape.resize(info.ndim);
    info.output_shape.resize(info.ndim);
    info.input_strides.resize(info.ndim);
    info.output_strides.resize(info.ndim);
    
    info.input_size = 1;
    info.output_size = 1;
    
    for (size_t i = 0; i < info.ndim; i++) {
        info.input_shape[i] = input_desc->shape()[i];
        info.input_strides[i] = input_desc->strides()[i];
        info.input_size *= input_desc->shape()[i];
        
        if (i == dim) {
            info.output_shape[i] = 1;
            info.output_strides[i] = output_desc->strides()[i];
        } else {
            info.output_shape[i] = input_desc->shape()[i];
            info.output_strides[i] = output_desc->strides()[i];
            info.output_size *= input_desc->shape()[i];
        }
    }
    
    *desc_ptr = new Descriptor(
        handle->device,
        handle->device_id,
        std::move(info),
        0
    );
    
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t Descriptor::calculate(
    void *workspace,
    size_t workspace_size,
    void *output,
    const void *input,
    void *stream) const {
    
    hipStream_t hip_stream = static_cast<hipStream_t>(stream);
    
    switch (_info.dtype) {
        case INFINI_DTYPE_F32:
            return ::launchReduceMeanKernel<float>(_info, output, input, hip_stream);
        case INFINI_DTYPE_F16:
            return ::launchReduceMeanKernel<fp16_t>(_info, output, input, hip_stream);
        case INFINI_DTYPE_BF16:
            return ::launchReduceMeanKernel<bf16_t>(_info, output, input, hip_stream);
        default:
            return INFINI_STATUS_BAD_TENSOR_DTYPE;
    }
}

} // namespace op::reduce_mean::nvidia