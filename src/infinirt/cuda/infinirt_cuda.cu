#include "../../utils.h"
#include "infinirt_cuda.cuh"
#include <hip/hip_runtime.h>

#define CHECK_CUDART(RT_API) CHECK_INTERNAL(RT_API, hipSuccess)

namespace infinirt::cuda {
infiniStatus_t getDeviceCount(int *count) {
    CHECK_CUDART(hipGetDeviceCount(count));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t setDevice(int device_id) {
    CHECK_CUDART(hipSetDevice(device_id));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t deviceSynchronize() {
    CHECK_CUDART(hipDeviceSynchronize());
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamCreate(infinirtStream_t *stream_ptr) {
    hipStream_t stream;
    CHECK_CUDART(hipStreamCreate(&stream));
    *stream_ptr = stream;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamDestroy(infinirtStream_t stream) {
    CHECK_CUDART(hipStreamDestroy((hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamSynchronize(infinirtStream_t stream) {
    CHECK_CUDART(hipStreamSynchronize((hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamWaitEvent(infinirtStream_t stream, infinirtEvent_t event) {
#ifdef ENABLE_ILUVATAR_API
    return INFINI_STATUS_NOT_IMPLEMENTED;
#else
    CHECK_CUDART(hipStreamWaitEvent((hipStream_t)stream, (hipEvent_t)event));
    return INFINI_STATUS_SUCCESS;
#endif
}

infiniStatus_t eventCreate(infinirtEvent_t *event_ptr) {
    hipEvent_t event;
    CHECK_CUDART(hipEventCreate(&event));
    *event_ptr = event;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventRecord(infinirtEvent_t event, infinirtStream_t stream) {
    CHECK_CUDART(hipEventRecord((hipEvent_t)event, (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventQuery(infinirtEvent_t event, infinirtEventStatus_t *status_ptr) {
    auto status = hipEventQuery((hipEvent_t)event);
    if (status == hipSuccess) {
        *status_ptr = INFINIRT_EVENT_COMPLETE;
    } else if (status == hipErrorNotReady) {
        *status_ptr = INFINIRT_EVENT_NOT_READY;
    } else {
        CHECK_CUDART(status);
    }
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventSynchronize(infinirtEvent_t event) {
    CHECK_CUDART(hipEventSynchronize((hipEvent_t)event));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventDestroy(infinirtEvent_t event) {
    CHECK_CUDART(hipEventDestroy((hipEvent_t)event));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mallocDevice(void **p_ptr, size_t size) {
    CHECK_CUDART(hipMalloc(p_ptr, size));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mallocHost(void **p_ptr, size_t size) {
    CHECK_CUDART(hipHostMalloc(p_ptr, size));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t freeDevice(void *ptr) {
    CHECK_CUDART(hipFree(ptr));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t freeHost(void *ptr) {
    CHECK_CUDART(hipHostFree(ptr));
    return INFINI_STATUS_SUCCESS;
}

hipMemcpyKind toCudaMemcpyKind(infinirtMemcpyKind_t kind) {
    switch (kind) {
    case INFINIRT_MEMCPY_H2D:
        return hipMemcpyHostToDevice;
    case INFINIRT_MEMCPY_D2H:
        return hipMemcpyDeviceToHost;
    case INFINIRT_MEMCPY_D2D:
        return hipMemcpyDeviceToDevice;
    case INFINIRT_MEMCPY_H2H:
        return hipMemcpyHostToHost;
    default:
        return hipMemcpyDefault;
    }
}

infiniStatus_t memcpy(void *dst, const void *src, size_t size, infinirtMemcpyKind_t kind) {
    CHECK_CUDART(hipMemcpy(dst, src, size, toCudaMemcpyKind(kind)));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t memcpyAsync(void *dst, const void *src, size_t size, infinirtMemcpyKind_t kind, infinirtStream_t stream) {
    CHECK_CUDART(hipMemcpyAsync(dst, src, size, toCudaMemcpyKind(kind), (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mallocAsync(void **p_ptr, size_t size, infinirtStream_t stream) {
    CHECK_CUDART(hipMallocAsync(p_ptr, size, (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t freeAsync(void *ptr, infinirtStream_t stream) {
    CHECK_CUDART(hipFreeAsync(ptr, (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}
} // namespace infinirt::cuda
